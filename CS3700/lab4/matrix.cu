#include "hip/hip_runtime.h"
// CS3700 Example matrix multpilcation using GPU

#include <stdio.h>
#include <math.h>
#include <sys/time.h>

#define TILE_WIDTH 2
#define WIDTH 6

// Kernel function execute by the device (GPU)
__global__ void
sum (float *d_a, float *d_b, float *d_c, const int n) {
   int col = blockIdx.x * blockDim.x + threadIdx.x ;
   int row = blockIdx.y * blockDim.y + threadIdx.y ;

   if (row < n && col < n) {
   	for (int i = 0; i<n; i ++){
	   d_c[row * n + i] = d_a[row * n + i ] + d_b[row * n + i] ;
	}
   }
}

// Kernel function execute by the device (GPU)
__global__ void
product (float *d_a, float *d_b, float *d_c, const int n) {
   int col = blockIdx.x * blockDim.x + threadIdx.x ;
   int row = blockIdx.y * blockDim.y + threadIdx.y ;

   float sum = 0;
   if (row < n && col < n) {
      for (int i = 0 ; i<n ; ++i) {
         sum += d_a[row * n + i ] * d_b[i * n + col] ;
      }
      d_c[row * n + col] = sum;
   }
}


// Utility function to print the input matrix
void printMatrix (float m[][WIDTH]) {
   int i, j;
   for (i = 0; i<WIDTH; ++i) {
      for (j = 0; j< WIDTH; ++j) {
         printf ("%d\t", (int)m[i][j]);
      }
      printf ("\n");
   }
}

// Main function execute by the host (CPU)
int main () {
   // host matrices
   float host_a[WIDTH][WIDTH],
         host_b[WIDTH][WIDTH],
         host_c[WIDTH][WIDTH];

   // device arrays
   float *device_a, *device_b, *device_c;

   int i, j;

   // initialize host matrices using random numbers
   time_t t;
   srand ((unsigned) time(&t));

   for (i = 0; i<WIDTH; ++i) {
      for (j = 0; j<WIDTH; j++) {
         host_a[i][j] = (float) (rand() % 50);
         host_b[i][j] = (float) (rand() % 50);
      }
   }

   printf ("Matrix A:\n");
   printMatrix (host_a);
   printf ("\n");

   printf ("Matrix B:\n");
   printMatrix (host_b);
   printf ("\n");

   // allocate device memory for input matrices
   size_t deviceSize = WIDTH * WIDTH * sizeof (float);
   hipMalloc ((void **) &device_a, deviceSize);
   hipMalloc ((void **) &device_b, deviceSize);

   // copy host matrices to device
   hipMemcpy (device_a, host_a, deviceSize, hipMemcpyHostToDevice );
   hipMemcpy (device_b, host_b, deviceSize, hipMemcpyHostToDevice );

   // allocate device memory to store computed result
   hipMalloc((void **) &device_c, deviceSize) ;

   dim3 dimBlock (WIDTH, WIDTH);
   dim3 dimGrid (WIDTH/TILE_WIDTH, WIDTH/TILE_WIDTH);
   product<<<dimGrid, dimBlock>>> (device_a, device_b, device_c, WIDTH);

   // copy result from device back to host
   hipMemcpy (host_c, device_c, deviceSize, hipMemcpyDeviceToHost);

   // output the computed result matrix
   printf ("A x B: \n");
   printMatrix (host_c);

   sum<<<dimGrid, dimBlock>>> (device_a, device_b, device_c, WIDTH);

   // copy result from device back to host
   hipMemcpy (host_c, device_c, deviceSize, hipMemcpyDeviceToHost);

   // output the computed result matrix
   printf ("\nA + B: \n");
   printMatrix (host_c);

   hipFree (device_a);
   hipFree (device_b);
   hipFree (device_c);
   return 0;
}
